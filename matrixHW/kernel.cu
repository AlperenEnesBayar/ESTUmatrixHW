#include "hip/hip_runtime.h"
﻿// alperen

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <chrono>
#include <iostream>


#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult(int* a, int* b, int* c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

__global__ void gpu_square_matrix_mult(int* d_a, int* d_b, int* d_result, int n)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if (idx >= n * n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if (idx >= n * n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows)
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}

void cpu_matrix_mult(int* h_a, int* h_b, int* h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h)
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}


int main(int argc, char const* argv[])
{
    int m, n, k;

    srand(3333);
    printf("Please enter this three parameters with a space between them:\n-> First matrices x dimension\n-> First matrices y and second matrices x dimension\n-> Second matrices y dimension\n");
    scanf("%d %d %d", &m, &n, &k);


    int* h_a, * h_b, * h_c, * h_cc;
    hipHostMalloc((void**)&h_a, sizeof(int) * m * n);
    hipHostMalloc((void**)&h_b, sizeof(int) * n * k);
    hipHostMalloc((void**)&h_c, sizeof(int) * m * k);
    hipHostMalloc((void**)&h_cc, sizeof(int) * m * k);

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
    }

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    auto start_time = std::chrono::high_resolution_clock::now();

    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, sizeof(int) * m * n);
    hipMalloc((void**)&d_b, sizeof(int) * n * k);
    hipMalloc((void**)&d_c, sizeof(int) * m * k);


    hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    if (m == n && n == k)
    {
        gpu_square_matrix_mult << <dimGrid, dimBlock >> > (d_a, d_b, d_c, n);
    }
    else
    {
        gpu_matrix_mult << <dimGrid, dimBlock >> > (d_a, d_b, d_c, m, n, k);
    }

    hipMemcpy(h_c, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    auto stop_time = std::chrono::high_resolution_clock::now();


    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    auto mine_gpu_elapsed_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(stop_time - start_time).count();
    std::cout << "Time elapsed on matrix multiplication of "<< m <<"x" << n << ", " << n << "x" << k <<" on GPU : " << mine_gpu_elapsed_time_ms <<" ms.\n\n";



    hipEventRecord(start, 0);
    auto start_time_cpu = std::chrono::high_resolution_clock::now();
    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);
    auto stop_time_cpu = std::chrono::high_resolution_clock::now();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    auto mine_cpu_elapsed_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(stop_time_cpu - start_time_cpu).count();
    std::cout << "Time elapsed on matrix multiplication of " << m << "x" << n << ", " << n << "x" << k << " on CPU : " << mine_cpu_elapsed_time_ms << " ms.\n\n";


    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            if (h_cc[i * k + j] != h_c[i * k + j])
            {
                all_ok = 0;
            }
        }
    }

    if (all_ok)
    {
        std::cout << "all results are correct!!!, speedup = " << mine_cpu_elapsed_time_ms / mine_gpu_elapsed_time_ms;
    }
    else
    {
        printf("incorrect results\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}
